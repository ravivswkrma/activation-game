#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */
#include<iostream>
#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"

using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/


//Traversing the level zero and increamenting the indegree of nodes connected to them
__global__ void traverseLevelZero(int *csr,int *offset, bool *isActive, int *aid, int *level, int vertices, int cnst, int currLvl)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id<vertices && level[id]==currLvl)
    {
        
        isActive[id]=true;

        aid[id]=0;

        int sidx = offset[id];
        int eidx = offset[id+1];

        for(int i=sidx; i<eidx; i++)
            atomicCAS(&level[csr[i]],-1,currLvl+1);
        

        for(int i=sidx; i<eidx; i++)
            atomicInc((unsigned int *)&aid[csr[i]], 15000); 
        
    }

}


__global__ void levelZero(int *apr, int *level, int v,int cnst)
{
    int id = blockIdx.x *blockDim.x + threadIdx.x;

    if(id<v && apr[id]==0)
        level[id]=0;    
}

//This kernel applies the activation rule i.e the first rule on nodes
__global__ void activatingVertices( int *offset, int *csr, int *level, int *aid, int *apr, bool *isActive,int V,int currLvl, int cnst, int edges)
{ 
    
    int id = blockIdx.x* blockDim.x + threadIdx.x;

    int sidx = offset[id];

    int eidx = offset[id+1];

    if(id < V && level[id] == currLvl)
    {
        for(int i=sidx; i<eidx; i++)
            atomicCAS(&level[csr[i]],-1,currLvl+1); 

        if(apr[id] <= aid[id])
            isActive[id]=true;   
        
    }
}
 //This kernel applies the deactivation rule i.e the first rule on nodesd   
__global__ void deactivatingVertices(int *offset, int *csr, int *level, int *aid, int *apr, bool *isActive,int V,int currLvl, int cnst, int edges)
{ 
    
    int id = blockIdx.x *blockDim.x + threadIdx.x;

    if(id<V && level[id]==currLvl)
    {
        //Apply Deactivation rule
        if((id-1)>=0 && (id+1)<V && level[id-1]==currLvl && level[id+1]==currLvl && isActive[id+1] !=true && isActive[id-1] != true )
            isActive[id]=false;
        
        //Checking whether the node is already active if active then increament the indegree of next level nodes which are connected to it
        if(isActive[id])
            for(int i=offset[id]; i<offset[id+1];i++)
                atomicInc((unsigned int*)&aid[csr[i]] ,  15000);
            
    }
}

//Applying activation and deactivation on the last level
__global__ void lastLevel(int *level, int *aid, int *apr, bool *isActive,int V,int currLvl, int cnst)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id<V && level[id]==currLvl)
    {
        // Apply activation rule
        if(aid[id]>= apr[id]){
            isActive[id]=true;   
      }

        //Apply Deactivation rule
        if((id-1)>=0 && (id+1)<V && level[id-1]==currLvl && level[id+1]==currLvl && isActive[id-1]!=true && isActive[id+1]!=true)
            isActive[id]=false;    
    }
}
    
//Finally calculating the result
__global__ void solve(bool *isActive, int *level, int *result,int vetex, int cnst) 
{  
    int id = blockIdx.x * blockDim.x + threadIdx.x;

        if(id<vetex && isActive[id])
            atomicInc((unsigned int *) &result[level[id]] , 15000);
}

/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    int *d_activeVertex;
    hipMalloc(&d_activeVertex, L*sizeof(int));
    hipMemset(d_activeVertex, 0, L*sizeof(int));

/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/

// Array for storing AID of each vertex.
hipMemset(d_aid, 0, V*sizeof(int));


//This array keeps track of whether the vetex is active or not
bool *d_isActive;
hipMalloc(&d_isActive, V*sizeof(bool));
hipMemset(d_isActive, 0, V*sizeof(bool));

//It stores the level of vetices
int *d_levelOfVertex; // We will inialize it to -1
hipMalloc(&d_levelOfVertex, V*sizeof(int));
hipMemset(d_levelOfVertex, -1, V*sizeof(int));

//No. of threads each block can have 
int blockSize = 512;

//Kernel Configuration starts
int numBlocks = (V+blockSize)/blockSize;
levelZero<<<numBlocks,blockSize>>>(d_apr,d_levelOfVertex, V, blockSize);
hipDeviceSynchronize();


//Now we know the level 0 nodes 


/*This kernel processes level 0*/
traverseLevelZero<<<numBlocks,blockSize>>>(d_csrList, d_offset, d_isActive, d_aid, d_levelOfVertex, V, blockSize, 0); 
hipDeviceSynchronize();

/*This kernel process level 1 to l-1 */

for(int i=1; i<L-1; i++)
{
    //It activates the vetices i.e applies rule 1
    activatingVertices<<<numBlocks,blockSize>>>(d_offset, d_csrList,d_levelOfVertex, d_aid, d_apr, d_isActive,V,i,blockSize, E); 
    hipDeviceSynchronize();


   //It deactivates the vetices i.e applies rule 2
    deactivatingVertices<<<numBlocks,blockSize>>>(d_offset, d_csrList,d_levelOfVertex, d_aid, d_apr, d_isActive,V,i,blockSize,E);
    hipDeviceSynchronize();
}

/* Last level is processed here */

lastLevel<<<numBlocks,blockSize>>>(d_levelOfVertex, d_aid, d_apr, d_isActive,V, L-1,blockSize); // send last level parameter
hipDeviceSynchronize();
       
/*This kernel calculates the final answer */
solve<<<numBlocks,blockSize>>>(d_isActive, d_levelOfVertex, d_activeVertex,V,blockSize); //level, active     
hipDeviceSynchronize();

/*device to host transfer*/
hipMemcpy(h_activeVertex, d_activeVertex, L*sizeof(int), hipMemcpyDeviceToHost);


/********************************END OF CODE AREA**********************************/


double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
